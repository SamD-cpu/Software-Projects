#include "hip/hip_runtime.h"
#include <random>
#include <chrono>
#include <thread>
#include <algorithm>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
 

#define SIZE 32

bool checkCorrectness(int* resultDevice, int* resultHost, int m, int k);

/***************************************************************
** Description: naive implmentation of Matrix Multiplication
**  
**
** Precondition: takes in values that factors in size relative
** to the hardware architreture.
**
**Postcondition: completed matrix with correct values
*****************************************************************
*/
__global__ void gpuMM(int* A, int* B, int* Res, int m, int n, int k)
{
	//using cuda builtin variables to return the 
	//full global thread  ID for multiplication matrix execution of rows and columns
	int Row = blockIdx.y * blockDim.y + threadIdx.y;
	int Col = blockIdx.x * blockDim.x + threadIdx.x;
	int resVal = 0;
	if (Col < k && Row < m)
	{
		for (int i = 0; i < n; i++)
		{
			resVal += A[Row * n + i] * B[i * k + Col];//fuction multiplication
		}
		Res[Row * k + Col] = resVal;// save resulting value
	}
}
/***************************************************************
** Description: tilled implmentation of Matrix Multiplication,
**inspired from techdarting nivida example, linked above
** 
** Precondition: takes in values that factors in size relative
** to the hardware architreture.
**
**Postcondition: completed matrix with correct values   
*****************************************************************
*/
__global__ void gpuBlockMM(int* deviceA, int* deviceB, int* deviceRes, int dim)
{
	__shared__ int tileA[SIZE][SIZE];
	__shared__ int tileB[SIZE][SIZE];
	int Row = blockIdx.y * SIZE + threadIdx.y;
	int Col = blockIdx.x * SIZE + threadIdx.x;

	int resVal = 0;

	for (int i = 0; i < gridDim.x; ++i)
	{
		if ((Row * dim + i * SIZE + threadIdx.x) <= dim * dim)
		{
			//copy elements to the shared memory for row load
			tileA[threadIdx.y][threadIdx.x] = deviceA[(Row * dim) + (i * SIZE) + threadIdx.x];
		}
		else
		{
			tileA[threadIdx.y][threadIdx.x] = 0;
		}
		if (((i * SIZE + threadIdx.y) * dim + Col) <= dim * dim)
		{
			//copy elements to the shared memory for column load
			tileB[threadIdx.y][threadIdx.x] = deviceB[(i * SIZE + threadIdx.y) * dim + Col];
		}
		else
		{
			tileB[threadIdx.y][threadIdx.x] = 0;
		}
		__syncthreads();//used when bounds of block reached

		for (int k = 0; k < SIZE; ++k)
		{
			//save values from multiplication
			resVal += tileA[threadIdx.y][k] * tileB[k][threadIdx.x];
		}
		__syncthreads();
	}
	if (Row < dim && Col < dim)// making sure the actual rows and columns do not exceed bounds
	{
		deviceRes[Row * dim + Col] = resVal;//saves resulting values
	}
}

/***************************************************************
** Description: implmentation of Matrix Multiplication,
** that will be run on the CPU for baseline comparison
**
** Precondition: takes in values that factors in size relative
** to the hardware architreture.
**
**Postcondition: completed matrix with correct values
*****************************************************************
*/

void cpuMM(int* hostA, int* hostB, int* hostRes, int m, int n, int k) {
	for (int i = 0; i < m; ++i)
	{
		for (int j = 0; j < k; ++j)
		{
			int C = 0.0;
			for (int l = 0; l < n; ++l)
			{
				C += hostA[i * n + l] * hostB[l * k + j];//saves results to index
			}
			hostRes[i * k + j] = C;//saves resulting values
		}
	}
}



int main()
{
	int m, n, k;
	srand(time(NULL));//using system internal clock for seed
	n = (double)1024;
	m = (double)1024;
	k = (double)1024;

	//allocates memory for host operation
	int* hostA, * hostB, * hostRes, * hostResCopy;
	hipHostMalloc((void**)& hostA, sizeof(int) * m * n);
	hipHostMalloc((void**)& hostB, sizeof(int) * n * k);
	hipHostMalloc((void**)& hostRes, sizeof(int) * m * k);
	hipHostMalloc((void**)& hostResCopy, sizeof(int) * m * k);

	//randomizes variables for host var
	for (int i = 0; i < m; ++i) {
		for (int j = 0; j < n; ++j) {
			hostA[i * n + j] = rand() % 20;
		}
	}

	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < k; ++j) {
			hostB[i * k + j] = rand() % 20;
		}
	}

	float gpuTimeMS, cpuTimeMS, memTimeMS;

//event call for calculating the execution time and memoryCopy
	hipEvent_t startEvent, stopEvent;
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	hipEvent_t startEvent2, stopEvent2;
	hipEventCreate(&startEvent2);
	hipEventCreate(&stopEvent2);

	hipEventRecord(startEvent2, 0);
	
	//allocates memory for device operation
	int* deviceA, * deviceB, * deviceRes;
	hipMalloc((void**)& deviceA, sizeof(int) * m * n);
	hipMalloc((void**)& deviceB, sizeof(int) * n * k);
	hipMalloc((void**)& deviceRes, sizeof(int) * m * k);

	// copy matrix A and B from host to device memory
	hipMemcpy(deviceA, hostA, sizeof(int) * m * n, hipMemcpyHostToDevice);
	hipMemcpy(deviceB, hostB, sizeof(int) * n * k, hipMemcpyHostToDevice);
	//consideration for a size that is not a multiple of thread per block
	dim3 dimGrid((k + SIZE - 1) / SIZE, (m + SIZE - 1) / SIZE);
	dim3 dimBlock(SIZE, SIZE);
	
	hipEventRecord(startEvent, 0);
	 //accounts for blocking preconditions
		//gpuBlockMM <<<dimGrid, dimBlock >>> (deviceA, deviceB, deviceRes, n);
	
		gpuMM << <dimGrid, dimBlock >> > (deviceA, deviceB, deviceRes, m, n, k);

	hipMemcpy(hostRes, deviceRes, sizeof(int) * m * k, hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);

	hipEventRecord(stopEvent2, 0);
	hipEventSynchronize(stopEvent2);

	hipEventElapsedTime(&memTimeMS, startEvent2, stopEvent2);
	printf("Size of M: %d ,Size of N: %d , Size of K: %d , Milliseconds for Memory recorded: %f , \nmatrix size %dx%d , threads %dx%d  GPU\n", m, n, k, memTimeMS, m,k,SIZE,SIZE);

	hipEventElapsedTime(&gpuTimeMS, startEvent, stopEvent);
	printf("Size of M: %d ,Size of N: %d , Size of K: %d , Milliseconds for Memory recorded: %f , \nmatrix size %dx%d , threads %dx%d  GPU\n", m, n, k, gpuTimeMS, m, k, SIZE, SIZE);

	hipEventRecord(startEvent, 0);

	cpuMM(hostA, hostB, hostResCopy, m, n, k);

	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	
	hipEventElapsedTime(&cpuTimeMS, startEvent, stopEvent);
	printf("Size of M: %d ,Size of N: %d , Size of K: %d , Milliseconds for Memory recorded: %f , \nmatrix size %dx%d , threads %dx%d  GPU\n", m, n, k, cpuTimeMS, m, k, SIZE, SIZE);

	checkCorrectness(hostResCopy, hostRes, m, k);
	//residual for both resulting array's
	double	sum_err = 0;
	for (int i = 0; i < m; ++i)
	{
		for (int j = 0; j < k; ++j)
		{
			sum_err += abs(hostResCopy[i * k + j] - hostRes[i * k + j]);
		}
	}
	std::cout << "average of residual:\n";
	std::cout << (double)sum_err / (double)(m * n) << "\n";

	// free memory
	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);
	hipEventDestroy(startEvent2);
	hipEventDestroy(stopEvent2);
	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceRes);
	hipHostFree(hostA);
	hipHostFree(hostB);
	hipHostFree(hostRes);
	hipHostFree(hostResCopy);
	return 0;
}

/***************************************************************
** Description: takes an input of two completed arrays
** used to check the correctness of matrix
**
** Precondition: takes in values that factors in size relative
** to the hardware architreture.
**
**Postcondition: returns a value that justifies the correctness of
** two matrices results
*****************************************************************
*/

bool checkCorrectness(int* resultDevice, int* resultHost, int m, int k)
{
	bool check = true;
	for (int i = 0; i < m; ++i)
	{
		for (int j = 0; j < k; ++j)
		{
			if (resultDevice[i * k + j] != resultHost[i * k + j])
			{
				printf("INCORRECT\n");
				return false;
			}
		}
	}
	if (check)
		printf("CORRECT\n");
	return true;
}
